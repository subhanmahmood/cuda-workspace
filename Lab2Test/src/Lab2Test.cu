/* Lab2Matrix.cu
 *
 *  Created on: 29 Feb 2020
 *      Author: sc01716
 */
#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>
#define BLOCK_SIZE 16

// Matrices are stored in row-major order
typedef struct {
	int width;
	int height;
	float* elements;
} Matrix;

__global__ void MatrixMultKern(const Matrix A, const Matrix B, const Matrix C) {
	// Calculate the column index of C and B
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	// Calculate the row index of C and of A
	int row = blockIdx.y * blockDim.y + threadIdx.y;

	if ((row < A.height) && (col < B.width)) {
		float Cvalue = 0;
		// each thread computes one element of the block sub-matrix
		for (int k = 0; k < A.width; ++k) {
			Cvalue += A.elements[row * A.width + k]
					* B.elements[k * B.width + col];
		}
		C.elements[row * C.width + col] = Cvalue;
	}
}

// Matrix multiplication -Host Code
//Matrix dimensions are assumed to be multiples of BLOCK_SIZE
void MatrixMult(const Matrix h_A, const Matrix h_B, Matrix h_C) {
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// Load A and B into device memory
	Matrix d_A;
	d_A.width = h_A.width;
	d_A.height = h_A.height;
	size_t size = h_A.width * h_A.height * sizeof(float);
	hipMalloc(&d_A.elements, size);
	hipMemcpy(d_A.elements, h_A.elements, size, hipMemcpyHostToDevice);

	Matrix d_B;
	d_B.width = h_B.width;
	d_B.height = h_B.height;
	size = h_B.width * h_B.height * sizeof(float);
	hipMalloc(&d_B.elements, size);
	hipMemcpy(d_B.elements, h_B.elements, size, hipMemcpyHostToDevice);

	// Allocate C in Device memory
	Matrix d_C;
	d_C.width = h_C.width;
	d_C.height = h_C.height;
	size = h_C.width * h_C.height * sizeof(float);
	hipMalloc(&d_C.elements, size);

	// Invoke Kernel
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid(d_B.width / dimBlock.x, d_A.height / dimBlock.y);
	hipEventRecord(start);
	MatrixMultKern<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);
	hipEventRecord(stop);

	// Read C from Device to Host
	hipMemcpy(h_C.elements, d_C.elements, size, hipMemcpyDeviceToHost);
	//printf("Copy h_C off device: %s\n", cudaGetErrorString(err));
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("Elapsed time was: %f\n milliseconds", milliseconds);
	// Free Device Memory
	hipFree(d_A.elements);
	hipFree(d_B.elements);
	hipFree(d_C.elements);
}

int main(int argc, char* argv[]) {
	Matrix A, B, C;
	// Read Dimensions of A and B
	A.height = atoi(argv[1]);
	A.width = atoi(argv[2]);
	B.height = A.width;
	B.width = atoi(argv[3]);

	A.elements = (float*) malloc(A.width * A.height * sizeof(float));
	B.elements = (float*) malloc(B.width * B.height * sizeof(float));
	C.height = A.height;
	C.width = B.width;
	C.elements = (float*) malloc(C.width * C.height * sizeof(float));

	for (int i = 0; i < A.height; i++)
		for (int j = 0; j < A.width; j++)
			A.elements[i * A.width + j] = (float) (rand() % 3);
	for (int i = 0; i < B.height; i++)
		for (int j = 0; j < B.width; j++)
			B.elements[i * B.width + j] = (float) (rand() % 2);
	MatrixMult(A, B, C);
	//printing full matrix
	/*
	 for (int i = 0; i < A.height; i++) {
	 for (int j = 0; j < A.width; j++)
	 printf("%f ", A.elements[i * A.width + j]);
	 printf("\n");
	 }
	 printf("\n");
	 for (int i = 0; i < B.height; i++) {
	 for (int j = 0; j < B.width; j++)
	 printf("%f ", B.elements[i * B.width + j]);
	 printf("\n");
	 }
	 printf("\n");
	 for (int i = 0; i < C.height; i++) {
	 for (int j = 0; j < C.width; j++)
	 printf("%f ", C.elements[i * C.width + j]);
	 printf("\n");
	 }

	 printf("\n");
	 */
	//printing only part of the matrix
	for (int i = 0; i < 16; i++) {
		for (int j = 0; j < 16; j++)
			printf("%f ", A.elements[i * A.width + j]);
		printf("\n");
	}
	printf("\n");
	for (int i = 0; i < 16; i++) {
		for (int j = 0; j < 16; j++)
			printf("%f ", B.elements[i * B.width + j]);
		printf("\n");
	}
	printf("\n");
	for (int i = 0; i < 16; i++) {
		for (int j = 0; j < 16; j++)
			printf("%f ", C.elements[i * C.width + j]);
		printf("\n");

	}

	printf("\n");
	printf("%d ",C.width);
	return 0;
}
